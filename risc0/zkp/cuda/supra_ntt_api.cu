#include "hip/hip_runtime.h"
#ifdef FEATURE_BABY_BEAR
# include <ff/baby_bear.hpp>
#endif

#include <ntt/ntt.cuh>

#ifndef __CUDA_ARCH__

extern "C"
RustError::by_value sppark_init()
{
    uint32_t lg_domain_size = 1;
    uint32_t domain_size = 1U << lg_domain_size;

    std::vector<fr_t> inout{domain_size};
    inout[0] = fr_t(1); inout[1] = fr_t(1);

    const gpu_t& gpu = select_gpu();

    try {
        NTT::Base(gpu, &inout[0], lg_domain_size, NTT::InputOutputOrder::NR,
                  NTT::Direction::forward, NTT::Type::standard);
        gpu.sync();
    } catch (const cuda_error& e) {
        gpu.sync();
    #ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
    #else
        return RustError{e.code()};
    #endif
    }

    return RustError{hipSuccess};
}

extern "C"
RustError::by_value batch_expand(fr_t* d_out, fr_t* d_in, uint32_t lg_domain_size,
                                 uint32_t lg_blowup, uint32_t poly_count)
{
    if (lg_domain_size == 0)
        return RustError{hipSuccess};

    uint32_t domain_size = 1U << lg_domain_size;
    uint32_t ext_domain_size = domain_size << lg_blowup;

    hipDeviceSynchronize();
    const gpu_t& gpu = select_gpu();

    // Determine the max power of 2 SM count
    size_t kernel_sms = gpu.sm_count();
    while (kernel_sms & (kernel_sms - 1))
        kernel_sms -= (kernel_sms & (0 - kernel_sms));

    try {
        for (size_t c = 0; c < poly_count; c++) {
            NTT::batch_expand(gpu, kernel_sms, &d_out[c * ext_domain_size],
                              &d_in[c * domain_size], lg_domain_size,
                              lg_blowup);
        }

        gpu.sync();
    } catch (const cuda_error& e) {
        gpu.sync();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
#else
        return RustError{e.code()};
#endif
    }

    return RustError{hipSuccess};
}

extern "C"
RustError::by_value batch_NTT(fr_t* d_inout, uint32_t lg_domain_size,
                              uint32_t poly_count)
{
    if (lg_domain_size == 0)
        return RustError{hipSuccess};

    uint32_t domain_size = 1U << lg_domain_size;

    hipDeviceSynchronize();
    const gpu_t& gpu = select_gpu();

    try {
        for (size_t c = 0; c < poly_count; c++) {
            NTT::Base_on_device(gpu, &d_inout[c * domain_size], lg_domain_size,
                                NTT::InputOutputOrder::RN,
                                NTT::Direction::forward, NTT::Type::standard);
        }

        gpu.sync();
    } catch (const cuda_error& e) {
        gpu.sync();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
#else
        return RustError{e.code()};
#endif
    }

    return RustError{hipSuccess};
}

extern "C"
RustError::by_value batch_iNTT(fr_t* d_inout, uint32_t lg_domain_size,
                               uint32_t poly_count)
{
    if (lg_domain_size == 0)
        return RustError{hipSuccess};

    uint32_t domain_size = 1U << lg_domain_size;

    hipDeviceSynchronize();
    const gpu_t& gpu = select_gpu();

    try {
        for (size_t c = 0; c < poly_count; c++) {
            NTT::Base_on_device(gpu, &d_inout[c * domain_size], lg_domain_size,
                                NTT::InputOutputOrder::NR,
                                NTT::Direction::inverse, NTT::Type::standard);
        }

        gpu.sync();
    } catch (const cuda_error& e) {
        gpu.sync();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
#else
        return RustError{e.code()};
#endif
    }

    return RustError{hipSuccess};
}

extern "C"
RustError::by_value batch_zk_shift(fr_t* d_inout, uint32_t lg_domain_size,
                                   uint32_t poly_count)
{
    if (lg_domain_size == 0)
        return RustError{hipSuccess};

    uint32_t domain_size = 1U << lg_domain_size;

    hipDeviceSynchronize();
    const gpu_t& gpu = select_gpu();

    try {
        for (size_t c = 0; c < poly_count; c++) {
            NTT::zk_shift(gpu, &d_inout[c * domain_size], lg_domain_size);
        }

        gpu.sync();
    } catch (const cuda_error& e) {
        gpu.sync();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
#else
        return RustError{e.code()};
#endif
    }

    return RustError{hipSuccess};
}

#endif
